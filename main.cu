#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>

int windowWidth = 2560;
int windowHeight = 1440;

GLuint pbo;
float globalZoom = 1.0f;
float offsetx = 0.0f;
float offsety = 0.0f;

void initPBO() {
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, windowWidth * windowHeight * 4, NULL, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
}

hipGraphicsResource* cudaPBOResource;

void registerPBOWithCUDA() {
    hipGraphicsGLRegisterBuffer(&cudaPBOResource, pbo, cudaGraphicsMapFlagsWriteDiscard);
}

__global__ void computeFractalKernel(uchar4* pixels, int width, int height, float zoom, float offsetX, float offsetY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    float jx = 1.5f * (x - width / 2) / (0.5f * zoom * width) + offsetX;
    float jy = (y - height / 2) / (0.5f * zoom * height) + offsetY;

    float zx = 0.0f;
    float zy = 0.0f;
    int iter = 0;
    const int maxIter = 2000;

    while (zx * zx + zy * zy < 4.0f && iter < maxIter) {
        float temp = zx * zx - zy * zy + jx;
        zy = 2.0f * zx * zy + jy;
        zx = temp;
        iter++;
    }

    uchar4 color;
    if (iter == maxIter) {
        color = make_uchar4(0, 0, 0, 255);
    } else {
        color = make_uchar4(iter % 256, iter % 256, iter % 256, 255);
    }

    int idx = y * width + x;
    pixels[idx] = color;
}

void launchFractalKernel(uchar4* d_pixels) {
    dim3 blockSize(16, 16);
    dim3 gridSize((windowWidth + blockSize.x - 1) / blockSize.x, (windowHeight + blockSize.y - 1) / blockSize.y);

    computeFractalKernel<<<gridSize, blockSize>>>(d_pixels, windowWidth, windowHeight, 1.0f, 0.0f, 0.0f);

    hipDeviceSynchronize();
}

void handleMouseWheel(int wheel, int direction, int x, int y) {
    float mouseNormX = 2.0f * (x / (float)windowWidth) - 1.0f;
    float mouseNormY = 1.0f - 2.0f * (y / (float)windowHeight);

    float oldFractalX = (mouseNormX / globalZoom) + offsetx;
    float oldFractalY = (mouseNormY / globalZoom) + offsety;

    if (direction > 0) {
        globalZoom *= 1.25f;
    } else if (direction < 0) {
        globalZoom *= 0.8f;
    }

    float newFractalX = (mouseNormX / globalZoom) + offsetx;
    float newFractalY = (mouseNormY / globalZoom) + offsety;

    offsetx += (oldFractalX - newFractalX);
    offsety += (oldFractalY - newFractalY);

    if (globalZoom < 1.0f) {
        globalZoom = 1.0f;
    }
}

void display() {
    hipGraphicsMapResources(1, &cudaPBOResource, 0);
    uchar4* d_pixels;
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void**)&d_pixels, &num_bytes, cudaPBOResource);

    launchFractalKernel(d_pixels);

    hipGraphicsUnmapResources(1, &cudaPBOResource, 0);

    glClear(GL_COLOR_BUFFER_BIT);
    glRasterPos2i(-1, -1);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glDrawPixels(windowWidth, windowHeight, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    glutSwapBuffers();
    glutPostRedisplay();
}

void cleanup() {
    if (cudaPBOResource) {
        hipGraphicsUnregisterResource(cudaPBOResource);
        cudaPBOResource = nullptr;
    }
    if (pbo) {
        glDeleteBuffers(1, &pbo);
        pbo = 0;
    }
    hipDeviceReset();
}

int main(int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(windowWidth, windowHeight);
    glutCreateWindow("CUDA Fractal");
    glutMouseWheelFunc(handleMouseWheel);

    GLenum err = glewInit();
    if (err != GLEW_OK) {
        std::cerr << "Error initializing GLEW: " << glewGetErrorString(err) << std::endl;
        return -1;
    }

    initPBO();

    registerPBOWithCUDA();

    glutDisplayFunc(display);

    atexit(cleanup);

    glutMainLoop();

    return 0;
}
